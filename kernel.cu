#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include "kernel.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include <thrust/gather.h>


__global__ void compute_vel(Fish* fishes, glm::vec2* vel2, unsigned int* grid_cell_indices, int* grid_cell_start, int* grid_cell_end,
    unsigned int N, BoidsParameters bp, unsigned int grid_size,
    double mouseX, double mouseY, bool mouse_pressed, AnimationVars av, int width, int height)
{
    const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) { return; }

    float xpos_avg = 0.0f, ypos_avg = 0.0f, xvel_avg = 0.0f, yvel_avg = 0.0f, neighboring_boids = 0.0f, close_dx = 0.0f, close_dy = 0.0f;

    Fish fish = fishes[index];

    // Find neighbours cells
    int cell_index = grid_cell_indices[index];
    int row_cells = width / (2 * bp.visionRange) + 1;
    int neighbour_cells[] = {cell_index, cell_index + 1, cell_index - 1, cell_index + row_cells, cell_index - row_cells, cell_index - row_cells - 1,
                    cell_index - row_cells + 1, cell_index + row_cells - 1, cell_index + row_cells + 1 };

    // Go through neighbour cells
    for (int j = 0; j < 9; ++j)
    {
        int current_cell = neighbour_cells[j];

        if (current_cell < 0 || current_cell >= grid_size)
            continue;

        // Iterate through fishes from neighbour cell
        for (int i = grid_cell_start[current_cell]; i < grid_cell_end[current_cell]; ++i)
        {
            if (i == index)
                continue;

            float dx = fish.x - fishes[i].x;
            float dy = fish.y - fishes[i].y;

            // Check if fish is in distance
            float distance = glm::sqrt(dx * dx + dy * dy);
            if (glm::abs(dx) < bp.visionRange && glm::abs(dy))
            {

                float distance2 = dx * dx + dy * dy;
                if (distance2 < bp.protectedRange)
                {
                    close_dx += fish.x - fishes[i].x;
                    close_dy += fish.y - fishes[i].y;
                }
                else if (distance2 < bp.visionRange * bp.visionRange)
                {
                    xpos_avg += fishes[i].x;
                    ypos_avg += fishes[i].y;
                    xvel_avg += fishes[i].vx;
                    yvel_avg += fishes[i].vy;

                    neighboring_boids += 1;
                }

            }
        }
    }

    if (neighboring_boids > 0)
    {
        xpos_avg = xpos_avg / neighboring_boids;
        ypos_avg = ypos_avg / neighboring_boids;
        xvel_avg = xvel_avg / neighboring_boids;
        yvel_avg = yvel_avg / neighboring_boids;

        // Add the centering / matching contributions to velocity
        fish.vx = (fish.vx + (xpos_avg - fish.x) * bp.cohesion + (xvel_avg - fish.vx) * bp.alignment);
        fish.vy = (fish.vy + (ypos_avg - fish.y) * bp.cohesion + (yvel_avg - fish.vy) * bp.alignment);

    }
    // Add the avoidance contribution to velocity
    fish.vx = fish.vx + (close_dx * bp.separation);
    fish.vy = fish.vy + (close_dy * bp.separation);

    // keep fishes in bounds
    if (fish.x < av.margin)
        fish.vx += av.turn_factor;
    if (fish.x > width - av.margin)
        fish.vx -= av.turn_factor;

    if (fish.y < av.margin)
        fish.vy += av.turn_factor;
    if (fish.y > height - av.margin)
        fish.vy -= av.turn_factor;

    // avoid coursor if mouse pressed
    if (mouse_pressed)
    {
        double x_diff = mouseX - fish.x;
        double y_diff = mouseY - fish.y;
        if (x_diff < av.margin && x_diff > -av.margin && y_diff < av.margin && y_diff > -av.margin)
        {
            if (x_diff > -av.margin && x_diff < 0)
                fish.vx += av.turn_factor;
            if (x_diff < av.margin && x_diff >= 0)
                fish.vx -= av.turn_factor;

            if(y_diff > -av.margin && y_diff < 0)
                fish.vy += av.turn_factor;
            if (y_diff < av.margin && y_diff >= 0)
                fish.vy -= av.turn_factor;
        }
    }

    // check if speed is < max_speed
    float speed = glm::sqrt(fish.vx * fish.vx + fish.vy * fish.vy);
    if (speed > av.max_speed)
    {
        fish.vx = (fish.vx / speed) * av.max_speed;
        fish.vy = (fish.vy / speed) * av.max_speed;
    }
    if (speed < av.min_speed)
    {
        fish.vx = (fish.vx / speed) * av.min_speed;
        fish.vy = (fish.vy / speed) * av.min_speed;
    }

    // update velocities
    vel2[index].x = fish.vx;
    vel2[index].y = fish.vy;
}

__global__ void update_pos_vel(Fish* fishes, glm::vec2* vel, unsigned int N, float speed_scale, int width, int height)
{
    const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) { return; }

    fishes[index].x += fishes[index].vx * speed_scale;
    fishes[index].y += fishes[index].vy * speed_scale;

    if (fishes[index].x < 0)
        fishes[index].x = 0;
    if (fishes[index].x > width)
        fishes[index].x = width;
    if (fishes[index].y < 0)
        fishes[index].y = 0;
    if (fishes[index].y > height)
        fishes[index].y = height;

    fishes[index].vx = vel[index].x;
    fishes[index].vy = vel[index].y;
}

__global__ void assign_grid_cell(Fish* fishes, unsigned int* grid_cells, unsigned int* indices, float cell_width, unsigned int N, int width)
{
    const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) { return; }

    float x = fishes[index].x;
    float y = fishes[index].y;

    int x_size = width / cell_width + 1;

    int x_cell = x / cell_width;
    int y_cell = y / cell_width;

    grid_cells[index] = y_cell * x_size + x_cell;
    indices[index] = index;
}

__global__ void compute_start_end_cell(unsigned int* grid_cell_indices, int* grid_cell_start, int* grid_cell_end, unsigned int N)
{
    const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) { return; }

    unsigned int grid_cell_id = grid_cell_indices[index];

    if (index == 0)
    {
        grid_cell_start[grid_cell_id] = 0;
        return;
    }
    unsigned int prev_grid_cell_id = grid_cell_indices[index - 1];
    if (grid_cell_id != prev_grid_cell_id)
    {
        grid_cell_end[prev_grid_cell_id] = index;
        grid_cell_start[grid_cell_id] = index;
        if (index == N - 1) 
        { 
            grid_cell_end[grid_cell_id] = index;
        }
    }
}

__global__ void copy_fishes_kernel(Fish* fishes, float* vertices, unsigned int N)
{
    const auto i = threadIdx.x + (blockIdx.x * blockDim.x);
    if (i >= N) { return; }

    Fish fish = fishes[i];

    vertices[i * 3 * 5] = fish.x;;
    vertices[i * 3 * 5 + 1] = fish.y;

    vertices[i * 3 * 5 + 2] = 0; vertices[i * 3 * 5 + 3] = 0; vertices[i * 3 * 5 + 4] = 0;

    vertices[i * 3 * 5 + 5] = fish.x - fish.lenght;
    vertices[i * 3 * 5 + 6] = fish.y + fish.width / 2;

    vertices[i * 3 * 5 + 7] = 0; vertices[i * 3 * 5 + 8] = 0; vertices[i * 3 * 5 + 9] = 0;

    vertices[i * 3 * 5 + 10] = fish.x - fish.lenght;
    vertices[i * 3 * 5 + 11] = fish.y - fish.width / 2;

    vertices[i * 3 * 5 + 12] = 0; vertices[i * 3 * 5 + 13] = 0; vertices[i * 3 * 5 + 14] = 0;
}
void CudaFish::initialize_simulation(unsigned int N, int width, int height)
{
    AnimationVars av();
    this->width = width;
    this->height = height;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
    }

    cudaStatus = hipMalloc(reinterpret_cast<void**>(&velocity_buffer), N * sizeof(glm::vec2));
    if(cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&grid_cell_indices), N * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&indices), N * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&fishes_gpu), N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&fishes_gpu_sorted), N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&vertices_array_gpu), 15 * N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    hipDeviceSynchronize();
}
void CudaFish::end_simulation()
{
    hipFree(velocity_buffer);
    hipFree(indices);
    hipFree(grid_cell_indices);
    hipFree(fishes_gpu);
    hipFree(fishes_gpu_sorted);
    hipFree(vertices_array_gpu);
}
void CudaFish::update_fishes(Fish* fishes, unsigned int N, BoidsParameters bp, double mouseX, double mouseY, bool mouse_pressed)
{
    hipError_t cudaStatus;
    
    const dim3 full_blocks_per_grid((N + av.block_size - 1) / av.block_size);
    const dim3 threads_per_block(av.block_size);

    float cell_width = 2 * bp.visionRange;
    unsigned int grid_size = (width / cell_width + 1) * (height / cell_width + 1);

    // Allocate memory for start and end indices
    int* grid_cell_start;
    int* grid_cell_end;

    cudaStatus = hipMalloc(reinterpret_cast<void**>(&grid_cell_start), grid_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&grid_cell_end), grid_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    // Copy data to gpu
    cudaStatus = hipMemcpy(fishes_gpu, fishes, N * sizeof(Fish), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    // Asign grid cell to every fish
    assign_grid_cell << <full_blocks_per_grid, threads_per_block >> > (fishes_gpu, grid_cell_indices, indices, cell_width, N, width);
    hipDeviceSynchronize();


    // Cast arrays to perform thrust operations
    auto thrust_gci = thrust::device_pointer_cast(grid_cell_indices);
    auto thrust_i = thrust::device_pointer_cast(indices);
    auto thrust_f = thrust::device_pointer_cast(fishes_gpu);
    auto thrust_fs = thrust::device_pointer_cast(fishes_gpu_sorted);

    // Sort fishes indicies by grid cell
    thrust::sort_by_key(thrust_gci, thrust_gci + N, thrust_i);

    // Compute start and end indices of grid cell
    compute_start_end_cell << <full_blocks_per_grid, threads_per_block >> > (grid_cell_indices, grid_cell_start, grid_cell_end, N);
    hipDeviceSynchronize();

    // Sort fish pos and vel by indices
    thrust::gather(thrust_i, thrust_i + N, thrust_f, thrust_fs);


    // Update velocity
    compute_vel << <full_blocks_per_grid, threads_per_block >> > (fishes_gpu_sorted, velocity_buffer, grid_cell_indices, grid_cell_start, grid_cell_end,
        N, bp, grid_size,
        mouseX, mouseY, mouse_pressed, av, width, height);
    hipDeviceSynchronize();


    // Update position
    update_pos_vel << <full_blocks_per_grid, threads_per_block >> > (fishes_gpu_sorted, velocity_buffer, N, bp.speed, width, height);
    hipDeviceSynchronize();
    

    // Copy data to CPU
    cudaStatus = hipMemcpy(fishes, fishes_gpu_sorted, N * sizeof(Fish), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    // Free memory
    hipFree(grid_cell_start);
    hipFree(grid_cell_end);
}
// Copy fishes to VBO
void CudaFish::copy_fishes(Fish* fishes, float* vertices_array, unsigned int N)
{
    hipError_t cudaStatus;

    const dim3 full_blocks_per_grid((N + av.block_size - 1) / av.block_size);
    const dim3 threads_per_block(av.block_size);

    // Copy data to gpu
    cudaStatus = hipMemcpy(fishes_gpu, fishes, N * sizeof(Fish), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    cudaStatus = hipMemcpy(vertices_array_gpu, vertices_array, 15 * N * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    copy_fishes_kernel << <full_blocks_per_grid, threads_per_block >> > (fishes_gpu, vertices_array_gpu, N);
    hipDeviceSynchronize();

    // Copy data to CPU
    cudaStatus = hipMemcpy(vertices_array, vertices_array_gpu, 15 * N * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

}